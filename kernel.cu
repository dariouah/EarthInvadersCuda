#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <conio.h>
#include <atomic>
#include <iostream>
#include <thread>
#include <chrono>
#include <hiprand/hiprand_kernel.h>
#include <windows.h>

#define RESET   "\033[0m"
#define GREEN   "\033[1m\033[32m"
#define ORANGE  "\033[38;5;208m"
#define BLUE    "\033[1m\033[34m"
#define PINK    "\033[1m\033[35m"
#define YELLOW  "\033[1m\033[33m"
#define RED     "\033[1m\033[31m"

// Configuracion grid
const int dimBlock = 16;

// Constantes de objetos
const char JUGADOR = 'W';
const char ALIEN = 'A';
const char NUBE = 'N';
const char CEFALOPODO = 'C';
const char DESTRUCTOR = 'D';
const char CRUCERO = 'R';
const char COMANDANTE = 'X';
const char MURO = 'B';

// Constantes de probabilidades
int ALIEN_PROB = 40;
int NUBE_PROB = 25;
int CEFALOPODO_PROB = 15;
int DESTRUCTOR_PROB = 5;
int CRUCERO_PROB = 13;
int COMANDANTE_PROB = 2;
const int MURO_PROB = 15;

// Conversion de probabilidades
// Con la generacion de numeros aleatorios, se compara con estas constantes para saber que objeto se genera
const int ALIEN_CONV = 40;
const int NUBE_CONV = 65;
const int CEFALOPODO_CONV = 80;
const int DESTRUCTOR_CONV = 85;
const int CRUCERO_CONV = 98;
const int COMANDANTE_CONV = 100;

// Constantes de puntajes
const int ALIEN_PUNT = 5;
const int NUBE_PUNT = 25;
const int CEFALOPODO_PUNT = 15;
const int DESTRUCTOR_PUNT = 5;
const int CRUCERO_PUNT = 13;
const int COMANDANTE_PUNT = 100;

// Constantes de otros
const int MURO_ALTURA = 5;
const int VIDA_INICIO = 5;
const int RONDA_TIEMPO = 2;
const int TAM_ONDA = 5;

using namespace std;

__global__ void generarTablero(char* tablero, int numFils, int numCols, unsigned int seed) {
    // Índice global del hilo
    int fila = (blockIdx.x * blockDim.x) + threadIdx.x;
    int columna = (blockIdx.y * blockDim.y) + threadIdx.y;
    int idx = fila * numCols + columna;

    // Comprobar que el índice esté dentro del tamaño del tablero
    if (fila < numFils && columna < numCols) {
        // Inicializar el generador de números aleatorios para este hilo con la semilla proporcionada
        hiprandState_t state;
        hiprand_init(seed + idx, idx, 0, &state);

        // Comprobamos si estamos en la fila adecuada para generar muros
        if (fila == (numFils - 5)) {
            // Generar un número aleatorio entre 0 y 99
            int rand_num = hiprand(&state) % 100;

            // Comprobar si se cumple la probabilidad de generar un muro
            if (rand_num < MURO_PROB) {
                // Contador para rastrear muros consecutivos
                __shared__ int murosSeguidos;
                murosSeguidos = 0;
                __syncthreads();

                // Generar muro o espacio vacío según las condiciones
                if (murosSeguidos < 3) {
                    tablero[idx] = 'B';
                    atomicAdd(&murosSeguidos, 1);
                }
                else {
                    tablero[idx] = ' ';
                }
            }
            else {
                tablero[idx] = ' '; // Espacio vacío si no se cumple la probabilidad
            }
        }
        else {
            tablero[idx] = ' '; // Espacio vacío en otras filas
        }
    }

}

// Función para mostrar el marco alrededor de la vida y la puntuación
void mostrarPuntuacion(int vida, int puntuacion, int numCols) {
    // Determinar el ancho del marco según el número de columnas del tablero
    int anchoMarco = numCols + 12; // El marco tendrá un ancho adicional de 12 caracteres

    // Mostrar el marco superior
    printf("+");
    for (int i = 0; i < anchoMarco - 2; ++i) {
        printf("-");
    }
    printf("+\n");

    // Mostrar la vida y la puntuación dentro del marco
    printf("| Vida: %3d    Puntuacion: %5d |\n", vida, puntuacion);

    // Mostrar el marco inferior
    printf("+");
    for (int i = 0; i < anchoMarco - 2; ++i) {
        printf("-");
    }
    printf("+\n");
}

// Función para mostrar el tablero
void mostrarTablero(char* tablero, int numFils, int numCols) {

    // Mostrar el tablero por CPU (solo es un hilo)
    for (int i = 0; i < numFils; i++) {
        for (int j = 0; j < numCols; j++) {
            char alien = tablero[i * numCols + j];
            switch (alien) {
            case 'A':
                printf(GREEN "[%c]" RESET, alien); // Alienígena – verde
                break;
            case 'N':
                printf(ORANGE "[%c]" RESET, alien); // Nube – naranja
                break;
            case 'C':
                printf(BLUE "[%c]" RESET, alien); // Cefalópodo – azul
                break;
            case 'D':
                printf(PINK "[%c]" RESET, alien); // Destructor – rosa
                break;
            case 'R':
                printf(YELLOW "[%c]" RESET, alien); // Crucero – amarillo
                break;
            case 'X':
                printf(RED "[%c]" RESET, alien); // Comandante – rojo
                break;
            default:
                printf("[%c]", alien); // Otros caracteres
            }
        }
        printf("\n");
    }
    printf("\n");
}


__global__ void reconversion(char* origenTablero, char* destinoTablero, int numFils, int numCols, unsigned int seed)
{
    int fila = (blockIdx.x * blockDim.x) + threadIdx.x;
    int columna = (blockIdx.y * blockDim.y) + threadIdx.y;
    int idx = fila * numCols + columna;
    hiprandState_t state;

    if (fila < numFils && columna < numCols) {
        char celdaOrigen = origenTablero[idx];
        // Clonamos el tablero en el destino
        destinoTablero[idx] = celdaOrigen;

        char resultado = 'N';
        int idxEncima = (fila - 1)* numCols + columna;
        int idxDebajo = idxDebajo = (fila + 1) * numCols + columna;
        int idxIzquierda = idxIzquierda = fila * numCols + (columna - 1);
        int idxDerecha = idxDerecha = fila * numCols + (columna + 1);

        // Comprobacion para hacer conversiones de alienigenas
        if (celdaOrigen == 'N' || celdaOrigen == 'A') {
            if (fila > 0 && fila < (numFils - 1) && columna>0 && columna < (numCols - 1)) {
                // Cambios el resultado por defecto si es una nube
                if (celdaOrigen == 'N') {
                    resultado = 'C';
                }
                if (origenTablero[idxEncima] == 'A' && origenTablero[idxDebajo] == 'A' && origenTablero[idxIzquierda] == 'A' && origenTablero[idxDerecha] == 'A') {
                    destinoTablero[idxEncima] = ' ';
                    destinoTablero[idxDebajo] = ' ';
                    destinoTablero[idxIzquierda] = ' ';
                    destinoTablero[idxDerecha] = ' ';
                    destinoTablero[idx] = resultado;

                }
            }
        } 
        else if (celdaOrigen == 'X') {
            hiprand_init(seed + idx, idx, 0, &state); // 1234 es una semilla, se puede cambiar
            if (fila > 0 && fila < (numFils - 1) && columna > 0 && columna < (numCols - 1)) {
                // Generar número aleatorio
                int aleatorio = hiprand(&state) % 100;
                if (aleatorio < 10) {
                    // Generar alienígenas en las posiciones adyacentes
                    destinoTablero[idxEncima] = ' ';
                    destinoTablero[idxDebajo] = ' ';
                    destinoTablero[idxIzquierda] = ' ';
                    destinoTablero[idxDerecha] = ' ';
                }
            }
        }
    }
}

__global__ void descenso(char* origenTablero, char* destinoTablero, int numFils, int numCols)
{
    int fila = (blockIdx.x * blockDim.x) + threadIdx.x;
    int columna = (blockIdx.y * blockDim.y) + threadIdx.y;
    int idx = fila * numCols + columna;

    if (fila < (numFils - 1) && columna < numCols) {
        char celdaOrigen = origenTablero[idx];
        char celdaDestino = origenTablero[idx + numCols];

        // Debemos de considerar los casos donde el destino sea un muro
        if (celdaDestino == 'B') {

            // Si la celda de origen es un destructor, indicamos que choca con muro
            if (celdaOrigen == 'D') {
                destinoTablero[idx + numCols] = 'd';

            }
            // Si la celda de origen es un crucero, rompe el muro
            else if (celdaOrigen == 'R') {
                destinoTablero[idx + numCols] = 'r';
            }
            // Si la celda de origen es un comandante, rompe el muro
            else if (celdaOrigen == 'X') {
                destinoTablero[idx + numCols] = 'X';
            }
            else {
                destinoTablero[idx + numCols] = 'B';
            }
            destinoTablero[idx + 2 * numCols] = ' ';
        }
        else if (celdaDestino == 'W') {
            
            if (celdaOrigen == ' ') {
                destinoTablero[idx + numCols] = 'W';
            }
            else {
                destinoTablero[idx + numCols] = '-';
			}
        }
        else {
            if (celdaOrigen != 'B') {
                destinoTablero[idx + numCols] = celdaOrigen;
            }
        }
    }
}

__global__ void generacion(char* origenTablero, char* destinoTablero, int numFils, int numCols, int* bolsaAleatorios)
{
    int fila = (blockIdx.x * blockDim.x) + threadIdx.x;
    int columna = (blockIdx.y * blockDim.y) + threadIdx.y;
    int idx = fila * numCols + columna;

    if (fila == 0 && columna < numCols) {
        int aleatorio = bolsaAleatorios[columna];
        if (aleatorio < ALIEN_CONV) {
            destinoTablero[idx] = ALIEN;
        }
        else if (aleatorio < NUBE_CONV) {
            destinoTablero[idx] = NUBE;
        }
        else if (aleatorio < CEFALOPODO_CONV) {
            destinoTablero[idx] = CEFALOPODO;
        }
        else if (aleatorio < DESTRUCTOR_CONV) {
            destinoTablero[idx] = DESTRUCTOR;
        }
        else if (aleatorio < CRUCERO_CONV) {
            destinoTablero[idx] = CRUCERO;
        }
        else if (aleatorio < COMANDANTE_CONV) {
            destinoTablero[idx] = COMANDANTE;
        }
        else {
            destinoTablero[idx] = ' ';
        }
    }
}

__global__ void desintegracion(char* origenTablero, char* destinoTablero, int numFils, int numCols, int* puntuacion, int* vida, unsigned int seed)
{
    int fila = (blockIdx.x * blockDim.x) + threadIdx.x;
    int columna = (blockIdx.y * blockDim.y) + threadIdx.y;
    int idx = fila * numCols + columna;
    hiprandState_t state;

    // Id de hilo pertenezca al tablero y no es la ultima fila
    if (fila < numFils && columna < numCols) {
        char celdaOrigen = origenTablero[idx];
        // Clonamos el tablero en el destino
        destinoTablero[idx] = celdaOrigen;

        // Id de hilo pertenece a ultima fila, actualiza puntuaciones
        if (fila == (numFils - 1)) {
            if (celdaOrigen == ALIEN) {
                atomicAdd(puntuacion, ALIEN_PUNT);
            }
            else if (celdaOrigen == NUBE) {
                atomicAdd(puntuacion, NUBE_PUNT);
            }
            else if (celdaOrigen == CEFALOPODO) {
                atomicAdd(puntuacion, CEFALOPODO_PUNT);
            }
            // Si el destructor hubiese chocado con jugador, seria 'd' y no contaría puntuación
            else if (celdaOrigen == DESTRUCTOR) {
                atomicAdd(puntuacion, DESTRUCTOR_PUNT);
            }
            else if (celdaOrigen == CRUCERO) {
                atomicAdd(puntuacion, CRUCERO_PUNT);
            }
            else if (celdaOrigen == COMANDANTE) {
                atomicAdd(puntuacion, COMANDANTE_PUNT);
                atomicAdd(vida, 1);
            }
            // Solo mantenemos el jugador
            if (celdaOrigen!='d' && celdaOrigen!='W' && celdaOrigen != '-') {
			    destinoTablero[idx] = ' ';
			}
            // Si fue golpeado en descenso, lo tomamos en cuenta
            else if (celdaOrigen == '-') {
				destinoTablero[idx] = 'W';
                atomicSub(vida, 1);
			}
        }

        // Si la celda es un destructor, generamos la onda expansiva
        if (celdaOrigen == 'd' || (celdaOrigen == 'D' && fila == numFils-1)) {
            // Inicio del cuadrado de onda expansiva
            int filaIni = fila - TAM_ONDA;
            int columnaIni = columna - TAM_ONDA;
            // Cuadrado de onda expansiva
            for (int i = filaIni; i < filaIni + (TAM_ONDA * 2 + 1); i++) {
				for (int j = columnaIni; j < columnaIni + (TAM_ONDA * 2 + 1); j++) {
                    // Comprobamos que la celda pertenezca al tablero
					if (i >= 0 && i < numFils && j >= 0 && j < numCols) {
                        char impacto = origenTablero[i * numCols + j];
                        // Si la onda golpea al jugador, se resta vida
                        if (impacto == 'W' || impacto == '-') {
                            atomicSub(vida, 1);
						}
                        // La onda no puede destruir ni muros, ni otros destructores y cruceros pendientes de desintegracion
                        else if (impacto != 'B' && impacto != 'd' && impacto != 'r' && impacto != '-') {
							destinoTablero[i * numCols + j] = ' ';
                        }
					}
				}
            }
            // Tras las explosiones, recuperamos los muros
            if (celdaOrigen == 'd') {
				destinoTablero[idx] = 'B';
            }
        }

        // Si la celda es un crucero, generamos la onda expansiva
        if (celdaOrigen == 'r' || (celdaOrigen == 'R' && fila == numFils - 1)) {
            hiprand_init(seed + idx, 0, 0, &state);
            int aleatorio = hiprand(&state) % 2;
            int ondaIni;
            int ondaFin;
            int ondaInc;

            // Inicio del cuadrado de onda expansiva
            if (aleatorio == 0) {
                ondaIni = fila * numCols;
				ondaFin = fila * (numCols + 1);
				ondaInc = 1;
			}
			else {
				ondaIni = columna;
				ondaFin = numFils * numCols + columna;
				ondaInc = numCols; 
            }

            // Recorremos la onda
            for (int i = ondaIni; i < ondaFin; i += ondaInc) {
                // Comprobamos que la celda pertenezca al tablero
                char impacto = origenTablero[i];
                // Si golpea al jugador, se pone como jugador dañado
                if (impacto == 'W' || impacto == '-') {
                    atomicSub(vida, 1);
                }
                // No tenemos en consideracion el muro porque puede romperlo
                else if (impacto != 'd' && impacto != 'r' && impacto != '-') {
                    destinoTablero[i] = ' ';
                }
            }

            // Tras las explosiones, desintegramos la nave
            if (celdaOrigen == 'r') {
                destinoTablero[idx] = ' ';  
            }
        }
    }
}

int main() {

    // Inicializamos por defecto
    int numFils = 15;
    int numCols = 10;
    char ejecucion = 'm';
    int puntuacion = 0;
    int vida = VIDA_INICIO;
    char caracter;
    int posJugador;
    bool incorrecto = false;
    // Inicializacion de la semilla para los numeros aleatorios
    srand(time(NULL));


    // Solicitar al usuario que ingrese los valores
    printf("+-------------------------------------------------+\n");
    printf("|               Configuracion Inicial             |\n");
    printf("+-------------------------------------------------+\n");
    printf("| Por favor, ingrese la configuracion inicial:    |\n");
    printf("|-------------------------------------------------|\n");
    printf("| Numero de filas (>=15): ");
    scanf("%d", &numFils);
    // Limpiar el búfer de entrada para evitar problemas
    while (getchar() != '\n');

    printf("| Numero de columnas (>=10): ");
    scanf("%d", &numCols);
    // Limpiar el búfer de entrada para evitar problemas
    while (getchar() != '\n');

    printf("| Modo de ejecucion (<m>, <a>): ");
    // Leer el carácter desde la entrada estándar
    scanf(" %c", &ejecucion);
    // Limpiar el búfer de entrada para evitar problemas
    while (getchar() != '\n');

    // Comprobamos los valores introducidos
    if (numFils < 15 || numCols < 10 || (ejecucion != 'm' && ejecucion != 'a')) {
        printf("+-------------------------------------------------+\n");
        printf("|              Configuracion Incorrecta           |\n");
        printf("+-------------------------------------------------+\n");
        printf("| Valores introducidos incorrectos. Se elegiran    \n");
        printf("| valores predeterminados:                         \n");
        printf("| Numero de filas: 15\n");
        printf("| Numero de columnas: 10\n");
        printf("| Modo de ejecucion: m\n");
        printf("+-------------------------------------------------+\n");

        // Valores por defecto
        numFils = 15;
        numCols = 10;
        ejecucion = 'm';
        incorrecto = true;
    }

    // Reservamos memoria en device para las matrices que usaremos en la GPU
    char* origenTablero;
    char* destinoTablero;
    hipMalloc((void**)&origenTablero, numFils * numCols * sizeof(char));
    hipMalloc((void**)&destinoTablero, numFils * numCols * sizeof(char));

    // Configuración de la cuadrícula y los bloques
    dim3 threadsPerBlock(dimBlock, dimBlock);
    int altoGrid = (numFils + threadsPerBlock.x - 1) / threadsPerBlock.x;
    int anchoGrid = (numCols + threadsPerBlock.y - 1) / threadsPerBlock.y;
    dim3 numBlocks(altoGrid, anchoGrid);

    // Mostrar mensaje decorado
    if (incorrecto) {
        printf("\n\n\n\n\n\n\n\n\n");
    } else {
		printf("\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n");
    }
    printf("+-------------------------------------------------+\n");
    printf("|            Presione Enter para continuar        |\n");
    printf("+-------------------------------------------------+\n");
    getchar(); // Esperar a que el usuario presione Enter


    // Inicializamos tablero que usaremos en la CPU
    char* tableroCPU;
    tableroCPU = new char[numFils * numCols];
    // Llamada al kernel generarTablero para origenTablero
    generarTablero << <numBlocks, threadsPerBlock >> > (origenTablero, numFils, numCols, time(NULL));
    // Copiamos el tablero a la CPU
    hipMemcpy(tableroCPU, origenTablero, numFils * numCols * sizeof(char), hipMemcpyDeviceToHost);

    // Añadimos posicion del jugador
    posJugador = numCols / 2;
    tableroCPU[(numFils - 1) * numCols + posJugador] = 'W';

    // Comenzamos el juego
    while (vida>0) {
        
        // Limpiamos la pantalla
        printf("\n\n\n\n\n\n\n\n\n");
        // Mostramos la vida y la puntuacion
        mostrarPuntuacion(vida, puntuacion, 22);
        // Mostramos el tablero
        mostrarTablero((char*)tableroCPU, numFils, numCols);

        //Aseguramos movimiento del jugador
        bool salir = false;
        // Buscamos pulsacion hasta que implique movimiento
        while (!salir) {
            // Comprobamos el modo de ejecucion
            if (ejecucion == 'm') {
                caracter = _getch();
            }
            else {
                // Generamos un movimiento aleatorio
                Sleep(1000);
                if ((rand() % 2) == 0) {
                    caracter = 'A';
                }
                else {
                    caracter = 'D';
                }  
            }

            if (caracter == 'a' || caracter == 'A') {
                salir = true;
                if (posJugador > 0) {
                    tableroCPU[(numFils - 1)*numCols+posJugador] = ' ';
                    posJugador--;
                    tableroCPU[(numFils - 1) * numCols + posJugador] = 'W';
                }
            }
            else if (caracter == 'd' || caracter == 'D') {
                salir = true;
                if (posJugador < numCols - 1) {
                    tableroCPU[(numFils - 1) * numCols + posJugador] = ' ';
                    posJugador++;
                    tableroCPU[(numFils - 1) * numCols + posJugador] = 'W';
                }
            }
        }

        // Actualizamos el tablero
        hipMemcpy(origenTablero, tableroCPU, numFils * numCols * sizeof(char), hipMemcpyHostToDevice);

        // Llamada al kernel reconversion
        reconversion << <numBlocks, threadsPerBlock >> > (origenTablero, destinoTablero, numFils, numCols, time(NULL));
        hipMemcpy(origenTablero, destinoTablero, numFils * numCols * sizeof(char), hipMemcpyDeviceToDevice);

        // Llamada al kernel descenso
        descenso << <numBlocks, threadsPerBlock >> > (origenTablero, destinoTablero, numFils, numCols);
        hipMemcpy(origenTablero, destinoTablero, numFils * numCols * sizeof(char), hipMemcpyDeviceToDevice);

        // Preparacion llamada al kernel generacion
        // Generamos un numero aleatorio para cada celda de la primera fila
        int* bolsaAleatoriosCPU;
        bolsaAleatoriosCPU = new int[numCols];
        for (int i = 0; i < numCols; i++) {
			bolsaAleatoriosCPU[i] = rand() % 100;
		}
        // Inicializamos el array de aleatorios en la GPU
        int* bolsaAleatoriosGPU;
        hipMalloc((void**)&bolsaAleatoriosGPU, numFils * numCols * sizeof(char));
        hipMemcpy(bolsaAleatoriosGPU, bolsaAleatoriosCPU, numFils * numCols * sizeof(char), hipMemcpyHostToDevice);
        // Llamada al kernel generacion
        generacion << <numBlocks, threadsPerBlock >> > (origenTablero, destinoTablero, numFils, numCols, bolsaAleatoriosGPU);
        hipMemcpy(origenTablero, destinoTablero, numFils * numCols * sizeof(char), hipMemcpyDeviceToDevice);

        // Preracion llamada al kernel desintegracion
        int* puntuacionGPU;
        int* vidaGPU;
        hipMalloc((void**)&puntuacionGPU, sizeof(int));
        hipMalloc((void**)&vidaGPU, sizeof(int));
        hipMemcpy(puntuacionGPU, &puntuacion, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(vidaGPU, &vida, sizeof(int), hipMemcpyHostToDevice);
        // Llamamos al kernel desintegracion
        desintegracion << <numBlocks, threadsPerBlock >> > (origenTablero, destinoTablero, numFils, numCols, puntuacionGPU, vidaGPU, time(NULL));
        hipMemcpy(origenTablero, destinoTablero, numFils * numCols * sizeof(char), hipMemcpyDeviceToDevice);

        // Mostramos la vida y la puntuacion
        hipMemcpy(&puntuacion, puntuacionGPU, sizeof(int), hipMemcpyDeviceToHost); // Copia la puntuacion al dispositivo
        hipMemcpy(&vida, vidaGPU, sizeof(int), hipMemcpyDeviceToHost); // Copia la vida al dispositivo
        // Comprobamos al final del ciclo
        hipMemcpy(tableroCPU, destinoTablero, numFils * numCols * sizeof(char), hipMemcpyDeviceToHost);
    }

    // Mostramos la vida y la puntuacion
    mostrarPuntuacion(vida, puntuacion, 22);
    // Mostramos el tablero
    mostrarTablero((char*)tableroCPU, numFils, numCols);


    printf("+-------------------------------------------------+\n");
    printf("|               HAS SIDO DERROTADO                |\n", puntuacion);
    printf("+-------------------------------------------------+\n");
    printf("|                PUNTUACION FINAL                 |\n", puntuacion);
    printf("|                     %5d                       |\n", puntuacion);
    printf("+-------------------------------------------------+\n");

    // Liberar memoria en device
    hipFree(origenTablero);
    hipFree(destinoTablero);


    return 0;
}